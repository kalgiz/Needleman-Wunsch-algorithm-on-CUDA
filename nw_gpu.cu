#include "hip/hip_runtime.h"
//Klaudia Algiz, 333811
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <climits>
#include <cstring>
#include <stdio.h>

using namespace std;

const int SEQ_SIZE = 60;
const int GP = 2;
int MIN_WEIGHT = 10;
const int KEYS_TO_SEARCH = 128;
const int TRIGRAMS_SIZE = 58;
const int TRIGRAMS_AMOUNT = 125;
const int BLOCK_DIM = KEYS_TO_SEARCH;
const int MAX_AT_TIME = 40000;

struct Seq
{
	char seq[SEQ_SIZE];
	long int nr;
	bool comple;
};


struct SeqPair
{
	int key[TRIGRAMS_SIZE];
	Seq value;
};

struct Edge
{
	long int nr1;
	long int nr2;
	int weight;
	bool compl1;
	bool compl2;
};


inline bool comparator (const SeqPair& seq1, const SeqPair& seq2)
{
	for (int i = 0; i < TRIGRAMS_SIZE; i++)
	{
		if (seq1.key[i] < seq2.key[i])
		{
			return true;
		}
		else if (seq1.key[i] > seq2.key[i])
		{
			return false;
		}
	}
	return false;
}

std::vector<Seq> buildComplementary(const std::vector<Seq>& fragments)
{
	std::vector<Seq> result;
	for (std::size_t i = 0; i < fragments.size(); i++)
	{
		Seq seq;
		for (int j = 0; j < SEQ_SIZE; j++)
		{
			switch( fragments[i].seq[j] )
			{
				case 'a':
					seq.seq[SEQ_SIZE-j-1] ='t';
					break;
				case 'c':
					seq.seq[SEQ_SIZE-j-1] = 'g';
					break;
				case 'g':
					seq.seq[SEQ_SIZE-j-1] = 'c';
					break;
				case 't':
					seq.seq[SEQ_SIZE-j-1] = 'a';
					break;
				case 'n':
					seq.seq[SEQ_SIZE-j-1] = 'n';
			}
		}
		seq.nr = fragments[i].nr;
		seq.comple = true;
		result.push_back(seq);
	}
	return result;
}

__global__ void sort_trigrams(char * seq, int * res, unsigned long fragmentsSize)
{
	int idx = blockIdx.x * BLOCK_DIM + threadIdx.x;
	if (idx >= fragmentsSize)
	{
		return;
	}
	
	int occs[TRIGRAMS_AMOUNT]; //how to use shared memory
	for (int i = 0; i < TRIGRAMS_AMOUNT; i++)
	{
		occs[i] = 0;
	}
	for (int i = 0; i < TRIGRAMS_SIZE; i++)
	{
		int vals[3];
		for (int j = 0; j < 3; j++)
		{
			switch(seq[idx*SEQ_SIZE+i+j])
			{
				case 'a':
					vals[j] = 0;
					break;
				case 'c':
					vals[j] = 1;
					break;
				case 'g':
					vals[j] = 2;
					break;
				case 't':
					vals[j] = 3;
					break;
				default:
					vals[j] = 4;
					break;
			}
		}
		int code = 25*vals[0] + 5*vals[1] + vals[2];
		occs[code]++;
	}
	
	//getting only existing trigrams
	int index = 0;
	for (int i = 0; i < TRIGRAMS_AMOUNT; i++)
	{
		for (int j = 0; j < occs[i]; j++)
		{
			res[idx*TRIGRAMS_SIZE+index] = i;
			index++;
		}
	}
	
	//bubble sort
	for (int i = 0; i < TRIGRAMS_SIZE-1; i++)
	{
		for (int j = i+1; j < TRIGRAMS_SIZE; j++)
		{
			if (occs[res[idx*TRIGRAMS_SIZE+j-1]] < 
				occs[res[idx*TRIGRAMS_SIZE+j]])
			{
				int tmp = res[idx*TRIGRAMS_SIZE+j-1];
				res[idx*TRIGRAMS_SIZE+j-1] = res[idx*TRIGRAMS_SIZE+j];
				res[idx*TRIGRAMS_SIZE+j] = tmp;
			}
			else if ((occs[res[idx*TRIGRAMS_SIZE+j-1]] == 
				occs[res[idx*TRIGRAMS_SIZE+j]]) && 
				(res[idx*TRIGRAMS_SIZE+j-1] > res[idx*TRIGRAMS_SIZE+j]))
			{
				int tmp = res[idx*TRIGRAMS_SIZE+j-1];
				res[idx*TRIGRAMS_SIZE+j-1] = res[idx*TRIGRAMS_SIZE+j];
				res[idx*TRIGRAMS_SIZE+j] = tmp;
			}
		}
	}
	
}

std::vector<SeqPair> sort_all_trigrams(std::vector<Seq> fragments)
{
	std::vector<SeqPair> result;
	char * input;
	char * devInput;
	int * output;
	int * devOutput;
	hipMalloc(&devInput, fragments.size()*SEQ_SIZE*sizeof(char));
	hipMalloc(&devOutput, fragments.size()*TRIGRAMS_SIZE*sizeof(int));
	hipHostMalloc(&input, fragments.size()*SEQ_SIZE*sizeof(char));
	hipHostMalloc(&output, fragments.size()*TRIGRAMS_SIZE*sizeof(int));
	for (int i = 0; i < fragments.size(); i++)
	{
		for (int j = 0; j < SEQ_SIZE; j++)
		{
			input[i*SEQ_SIZE+j] = fragments[i].seq[j];
		}
	}
	hipMemcpy(devInput, input, fragments.size()*SEQ_SIZE*sizeof(char), 
							hipMemcpyHostToDevice);
	int blocksNr = fragments.size()/BLOCK_DIM + (fragments.size()%BLOCK_DIM == 0 ? 0 : 1);
	sort_trigrams<<< blocksNr, BLOCK_DIM >>>(devInput, devOutput, fragments.size());
	hipMemcpy(output, devOutput, fragments.size()*TRIGRAMS_SIZE*sizeof(int), 
						hipMemcpyDeviceToHost);
	for (int i = 0; i < fragments.size(); i++)
	{
		SeqPair sp;
		sp.value = fragments[i];
		for (int j = 0; j < TRIGRAMS_SIZE; j++)
		{
			sp.key[j] = output[i*TRIGRAMS_SIZE+j];
		}
		result.push_back(sp);
	}
	hipFree(devInput);
	hipFree(devOutput);
	hipHostFree(input);
	hipHostFree(output);
	return result;
}

__global__ void countDistances(Seq * fragments, Edge * graph, 
										unsigned long fragmentsSize, int loopNr)
{
	int idx1 = blockIdx.x;
	__shared__ Seq seq1;
	if (threadIdx.x == 0)
	{
		seq1.nr = fragments[idx1].nr;
		seq1.comple = fragments[idx1].comple;
		for (int i = 0; i < SEQ_SIZE; i++)
			seq1.seq[i] = fragments[idx1].seq[i];
	}
	syncthreads();
	int idx2 = idx1 + threadIdx.x+1;
	if (idx1 >= fragmentsSize || idx2 >= fragmentsSize)
	{
		Edge * edge = graph + blockIdx.x*BLOCK_DIM+threadIdx.x;
		edge->weight = -1;
		return;
	}
	Seq *seq2 = fragments+idx2;
	
	int A[SEQ_SIZE+1][SEQ_SIZE+1];
 	// initialization
	for (int i = 0; i <= SEQ_SIZE; i++)
	{
		A[i][0] = 0;
		A[0][i] = 0;
	}
 		
	for (int i = 1; i <= SEQ_SIZE; i++)
		for (int j = 1; j <= SEQ_SIZE; j++)
		{
			if (seq1.seq[i-1] == seq2->seq[j-1]) A[i][j] = 1;
			else A[i][j] = -2;
		}
 		
 	// 	computing matrix
 		for (int i = 1; i <= SEQ_SIZE; i++)
		{
 			for (int j = 1; j <= SEQ_SIZE; j++)
 			{
 				A[i][j] = max(A[i-1][j-1] + A[i][j], max(
 					A[i][j-1]-GP, A[i-1][j]-GP) );
 			}
		}
		
 		int maks = INT_MIN;
		for (int i = 0; i <= SEQ_SIZE; i++)
		{
			maks = max(A[SEQ_SIZE][i], maks);
			maks = max(A[i][SEQ_SIZE], maks);
		}
 		Edge * edge = graph + blockIdx.x*BLOCK_DIM+threadIdx.x;
 		edge->nr1 = seq1.nr;
		edge->nr2 = seq2->nr;
		edge->compl1 = seq1.comple;
		edge->compl2 = seq2->comple;
		edge->weight = maks;
}

void createGraph(std::vector<Seq> fragments)
{
	Seq * input;
	Seq * devInput;
	Edge * output;
	Edge * devOutput;
	int graphEdgeNr = ((fragments.size()+MAX_AT_TIME-1)/MAX_AT_TIME)*MAX_AT_TIME*KEYS_TO_SEARCH;
	hipHostMalloc(&input, fragments.size()*sizeof(Seq));
	hipHostMalloc(&output, graphEdgeNr*sizeof(Edge));
	hipMalloc(&devInput, (MAX_AT_TIME+KEYS_TO_SEARCH)*sizeof(Seq));
	hipMalloc(&devOutput, MAX_AT_TIME*KEYS_TO_SEARCH*sizeof(Edge));
	for (int i = 0; i < fragments.size(); i++)
	{
		input[i] = fragments[i];
	}
	
	for (int i = 0; i < (fragments.size()+MAX_AT_TIME-1)/MAX_AT_TIME; i++)
	{
		int seqsToCopy = min(MAX_AT_TIME+KEYS_TO_SEARCH, (int)(fragments.size()-i*MAX_AT_TIME));
		hipMemcpy(devInput, input+i*MAX_AT_TIME, seqsToCopy*sizeof(Seq),
				hipMemcpyHostToDevice);
		int blocksNr = MAX_AT_TIME;
		countDistances<<< blocksNr, BLOCK_DIM >>>(devInput, devOutput, fragments.size(), i);
		hipMemcpy(output+i*MAX_AT_TIME*KEYS_TO_SEARCH, devOutput, MAX_AT_TIME*KEYS_TO_SEARCH*sizeof(Edge), 
				hipMemcpyDeviceToHost);
	}
	
	for (int i = 0; i < graphEdgeNr; i++)
	{
		if (output[i].weight >= MIN_WEIGHT && output[i].weight != -1)
		{
			cout << output[i].nr1 << "; " << output[i].nr2 << "; "
					<< output[i].weight << "; "
					<< (output[i].compl1 ? "1" : "0") << ", "
					<< (output[i].compl2 ? "1" : "0") << ";\n";
		}
	}

	hipFree(devInput);
	hipFree(devOutput);
	hipHostFree(input);
	hipHostFree(output);
}

std::vector<Seq> readEntrance()
{
	std::vector<Seq> result;
	std::string prev;
	long int prevNumber = 0;
	for (std::string line; std::getline(std::cin, line);) {
		if (line.size() > 0 && line[0] == '>')
		{
			if (prev != "") 
			{
				Seq seq;
				strcpy(seq.seq, prev.c_str());
				seq.nr = prevNumber;
				seq.comple = false;
				result.push_back(seq);
			}
			prev = "";
			prevNumber = atoi(line.substr(3).c_str());
		}
		else 
			prev += line;
	}
	Seq seq;
	strcpy(seq.seq, prev.c_str());
	seq.nr = prevNumber;
	seq.comple = false;
	result.push_back(seq);
	return result;
}

int main (int argc, char* argv[]) {
	if (argc != 1 && argc != 2)
	{
		std::cout << "wrong number of parameters\n";
		return 0;
	}
	if (argc == 2)
		MIN_WEIGHT = atoi(argv[1]);
	
	std::vector<Seq> fragments = readEntrance();

	std::vector<Seq> complFr = buildComplementary(fragments);
	fragments.insert(fragments.end(), complFr.begin(), complFr.end());
	
	std::vector<SeqPair> seqMap = sort_all_trigrams(fragments);
	std::sort(seqMap.begin(), seqMap.end(), comparator);
	fragments.clear();
	for (int i = 0; i < seqMap.size(); i++)
	{
		fragments.push_back(seqMap[i].value);
	}
	createGraph(fragments);
	
  return 0;
}